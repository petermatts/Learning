#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "../common/book.h"

#define N (33*1024)

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev1, *dev2, *dev3;

    HANDLE_ERROR(hipMalloc((void**) &dev1, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**) &dev2, N*sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**) &dev3, N*sizeof(int)));

    for(int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i;
    }

    HANDLE_ERROR(hipMemcpy(dev1, a, N*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev2, b, N*sizeof(int), hipMemcpyHostToDevice));

    add<<<128,128>>>(dev1, dev2, dev3);

    HANDLE_ERROR(hipMemcpy(c, dev3, N*sizeof(int), hipMemcpyDeviceToHost));

    for(int i = 0; i < N; i++) {
        if((a[i] + b[i]) != c[i]) {
            printf("Error: %d + %d != %d", a[i], b[i], c[i]);
        } else {
            printf("%d + %d = %d\n", a[i], b[i], c[i]);
        }
    }   

    hipFree(dev1);
    hipFree(dev2);
    hipFree(dev3);

    return 0;    
}