#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "../common/book.h"

#define sum_squares(x)  (x*(x+1)*(2*x+1)/6)

const int N = 10;
const int threadsPerBlock = 256;
const int blocksPerGrid = min(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float sum = 0;
    while (tid < N) {
        sum += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = sum;

    __syncthreads(); // sync threads in current block (shared mem is stored at block level)

    // log reduction of array - wicked cool and fast
    // threadsPerBlock needs to be a power of 2 for this
    // ? maybe it doesn't NEED to be if this is revised?
    int i = blockDim.x / 2;
    while(i != 0) {
        if(cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main(void) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    // allocate CPU memory
    a = (float *) malloc(N*sizeof(float));
    b = (float *) malloc(N*sizeof(float));
    partial_c = (float *) malloc(blocksPerGrid * sizeof(float));

    // allocate GPU memory
    HANDLE_ERROR(hipMalloc((void **) &dev_a, N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **) &dev_b, N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **) &dev_partial_c, blocksPerGrid * sizeof(float)));

    // fill CPU variables with values
    for(int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    // copy variable data from CPU to allocated memory on GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

    // do the dot product computation on the GPU kernel
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    // copy results back over to the CPU
    HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

    // verify results
    c = 0;
    for(int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    // printf( "Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares( (float)(N - 1) ) );
    printf("GPU gets %0.6g\n", c);
    printf("CPU gets %0.6g\n", sum_squares((float) (N-1)));

    // free GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    // free CPU memory
    free(a);
    free(b);
    free(partial_c);
}